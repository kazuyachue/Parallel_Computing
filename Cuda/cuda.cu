//
// http://forums.nvidia.com/index.php?showtopic=34309
//

#include <hip/hip_runtime.h>
#include <stdio.h>

// called from host, run on device
__global__ void add_arrays_gpu(float *in1,float *in2,float *out)
{
	int idx=threadIdx.x; // flat model
	
	out[idx]=in1[idx]+in2[idx];
}
int main()
{
	// pointers to host memory
	float *a,*b,*c;

	// pointers to device memory
	float *a_d,*b_d,*c_d;
	int N=18;
	int i;

	// allocate arrays a, b and c on host
	a=(float*)malloc(N*sizeof(float));
	b=(float*)malloc(N*sizeof(float));
	c=(float*)malloc(N*sizeof(float));

	// allocate arrays a_d, b_d and c_d on device
	hipMalloc((void**)&a_d,sizeof(float)*N);
	hipMalloc((void**)&b_d,sizeof(float)*N);
	hipMalloc((void**)&c_d,sizeof(float)*N);

	// initialize arrays a and b
	for(i=0;i<N;i++){
		a[i]= (float) i*i;
		b[i]=-(float) i/2.0f;
	}

	// copy input from host memory to device memory
	hipMemcpy(a_d,a,sizeof(float)*N,hipMemcpyHostToDevice);
	hipMemcpy(b_d,b,sizeof(float)*N,hipMemcpyHostToDevice);

	// execution configuration: How the threads are arranged, FLAT and LINEAR.
	dim3 dimGrid(1),dimBlock(N);
	add_arrays_gpu<<<dimGrid,dimBlock>>>(a_d, b_d, c_d);
	
	// copy result from device memory to host memory
	hipMemcpy(c,c_d,sizeof(float)*N,hipMemcpyDeviceToHost);
	
	for(i=0;i<N;i++)
		printf("c[%d]=%f\n",i,c[i]);
	
	free(a);
	free(b);
	free(c);
	hipFree(a_d);
	hipFree(b_d);
	hipFree(c_d);
}
